
#include <hip/hip_runtime.h>
// Try this with BUFSIZE <= 128
#define BUFSIZE 256

extern "C" __global__ void kernel() {
    __shared__ unsigned int buffer[BUFSIZE];

    // spin for a while...
    for (unsigned long long int i = 0; i < 1000ULL; i++) {
        __nanosleep(2000000U);
    }

    // non-trivial shared memory usage so it doesn't get optimized away
    buffer[threadIdx.x % BUFSIZE] = 1;
    unsigned int ans = buffer[(threadIdx.x + 1) % BUFSIZE];

    __nanosleep(ans);
}
